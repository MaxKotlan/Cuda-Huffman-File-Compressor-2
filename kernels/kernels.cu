#include "hip/hip_runtime.h"
#include "shared.h"
#include "kernels.h"

bool initalized = false;

__global__ void CalculateByteFrequency(Node* hashmap, const unsigned char* filebuffer, unsigned int filebufferSize ){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < filebufferSize) {
    
        unsigned char byte = filebuffer[idx];
        //hashmap[byte].character = byte;
        //hashmap[byte].frequency++;

    }
}


unsigned char* _device_buffer;
Node*  _device_hashmap;

void Init(unsigned int bufferSize, FrequencyMap& hashmap){
    gpuErrchk(hipMalloc((void **)&_device_buffer,                   bufferSize));
    gpuErrchk(hipMalloc((void **)&_device_hashmap, sizeof(Node)*hashmap.size()));
    gpuErrchk(hipMemcpy(_device_hashmap, hashmap.data(), sizeof(Node)*hashmap.size(), hipMemcpyHostToDevice));
    initalized = true;
}

void CudaGetCharacterFrequencies(FrequencyMap& hashmap, const std::vector<unsigned char>& buffer){
    if (!initalized) Init(buffer.size(), hashmap);
    gpuErrchk(hipMemcpy(_device_buffer, buffer.data(), buffer.size(), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    hipEventRecord(start);
    CalculateByteFrequency<<< buffer.size() / 1024 + 1, 1024 >>>( hashmap.data(), buffer.data(), buffer.size() );
    hipEventRecord(stop);
    gpuErrchk(hipEventSynchronize(stop));

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Cuda Kernel Execution took " << milliseconds << std::endl;
    gpuErrchk(hipMemcpy(hashmap.data(), _device_hashmap, sizeof(Node)*hashmap.size(), hipMemcpyDeviceToHost));
}