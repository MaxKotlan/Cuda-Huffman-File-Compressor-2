#include "hip/hip_runtime.h"
#include "shared.h"
#include "kernels.h"

bool initalized = false;

__global__ void CalculateByteFrequency(Node* hashmap, const unsigned char* filebuffer, unsigned int filebufferSize, unsigned int bufferIteration, unsigned int fileSize){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < filebufferSize && (idx + bufferIteration*filebufferSize) < fileSize) {
        unsigned char byte = filebuffer[idx];
        atomicAdd(&hashmap[byte].frequency, 1);
    }
}


unsigned char* _device_buffer;
Node*  _device_hashmap;

void Init(unsigned int bufferSize, FrequencyMap& hashmap){
    gpuErrchk(hipMalloc((void **)&_device_buffer,                         bufferSize));
    gpuErrchk(hipMalloc((void **)&_device_hashmap,       sizeof(Node)*hashmap.size()));
    gpuErrchk(hipMemcpy(_device_hashmap, hashmap.data(), sizeof(Node)*hashmap.size(), hipMemcpyHostToDevice));
    initalized = true;
}

void CudaGetCharacterFrequencies(FrequencyMap& hashmap, const std::vector<unsigned char>& buffer, unsigned int bufferIteration, unsigned int fileSize){
    if (!initalized) Init(buffer.size(), hashmap);
    gpuErrchk(hipMemcpy(_device_buffer, buffer.data(), buffer.size(), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);    

    hipEventRecord(start);
    //std::cout << "Initalizing kernel to calculate hashmap on " << hashmap.size() << " elements each of " << sizeof(Node)*hashmap.size() << std::endl;

    CalculateByteFrequency<<< buffer.size() / 1024 + 1, 1024 >>>( _device_hashmap, _device_buffer, buffer.size(), bufferIteration, fileSize);
    hipEventRecord(stop);
    gpuErrchk(hipEventSynchronize(stop));

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    /*If entire file has been read*/
    if ((bufferIteration+1)*buffer.size() >= fileSize){
        gpuErrchk(hipMemcpy(hashmap.data(), _device_hashmap, sizeof(Node)*hashmap.size(), hipMemcpyDeviceToHost));
        hipFree(_device_hashmap); hipFree(_device_buffer);

        /*In serial set characters in hashmap*/
        for (int i =0; i < hashmap.size(); i++)
            hashmap[i].character = (unsigned char)i;

        initalized = false;
    }
}